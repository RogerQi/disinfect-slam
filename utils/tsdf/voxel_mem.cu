#include "hip/hip_runtime.h"
#include <cassert>

#include "utils/cuda/errors.cuh"
#include "utils/tsdf/voxel_mem.cuh"

__global__ static void heap_init(int *heap) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < NUM_BLOCK) {
    heap[idx] = idx;
  }
}

__device__ __host__ Vector3<short> point2block(const Vector3<short> &point) {
  return point >> BLOCK_LEN_BITS;
}

__device__ __host__ Vector3<short> point2offset(const Vector3<short> &point) {
  return point & (BLOCK_LEN - 1);
}

__device__ __host__ unsigned int offset2index(const Vector3<short> &point_offset) {
  return point_offset.x + point_offset.y * BLOCK_LEN + point_offset.z * BLOCK_AREA;
}

VoxelMemPool::VoxelMemPool() {
  // initialize free block counter
  CUDA_SAFE_CALL(hipMallocManaged(&num_free_blocks_, sizeof(int)));
  *num_free_blocks_ = NUM_BLOCK;
  // intialize voxel data buffer
  CUDA_SAFE_CALL(hipMalloc(&voxels_rgbw_, sizeof(VoxelRGBW) * NUM_BLOCK * BLOCK_VOLUME));
  CUDA_SAFE_CALL(hipMalloc(&voxels_tsdf_, sizeof(VoxelTSDF) * NUM_BLOCK * BLOCK_VOLUME));
  CUDA_SAFE_CALL(hipMalloc(&voxels_segm_, sizeof(VoxelSEGM) * NUM_BLOCK * BLOCK_VOLUME));
  // initialize heap array
  CUDA_SAFE_CALL(hipMalloc(&heap_, sizeof(int) * NUM_BLOCK));
  heap_init<<<NUM_BLOCK / 256, 256>>>(heap_);
  CUDA_CHECK_ERROR;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void VoxelMemPool::ReleaseMemory() {
  CUDA_SAFE_CALL(hipFree(voxels_rgbw_));
  CUDA_SAFE_CALL(hipFree(voxels_tsdf_));
  CUDA_SAFE_CALL(hipFree(voxels_segm_));
  CUDA_SAFE_CALL(hipFree(num_free_blocks_));
  CUDA_SAFE_CALL(hipFree(heap_));
}

__device__ int VoxelMemPool::AquireBlock() {
  const int idx = atomicSub(num_free_blocks_, 1);
  assert(idx >= 1);

  const VoxelBlock block(heap_[idx - 1]);

  #pragma unroll
  for (int i = 0; i < BLOCK_VOLUME; ++i) {
    VoxelRGBW &voxel_rgbw = GetVoxel<VoxelRGBW>(i, block);
    VoxelTSDF &voxel_tsdf = GetVoxel<VoxelTSDF>(i, block);
    VoxelSEGM &voxel_segm = GetVoxel<VoxelSEGM>(i, block);
    voxel_rgbw.weight = 0;
    voxel_tsdf.tsdf = 1;
    voxel_segm.probability = .1;
  }

  return block.idx;
}

__device__ void VoxelMemPool::ReleaseBlock(const int block_idx) {
  const int idx = atomicAdd(num_free_blocks_, 1);
  assert(idx < NUM_BLOCK);

  heap_[idx] = block_idx;
}

__device__ __host__ int VoxelMemPool::NumFreeBlocks() const {
  return *num_free_blocks_;
}

