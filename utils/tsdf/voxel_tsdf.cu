#include "hip/hip_runtime.h"
#include "utils/cuda/arithmetic.cuh"
#include "utils/cuda/errors.cuh"
#include "utils/tsdf/voxel_tsdf.cuh"

#define MAX_IMG_H     1920
#define MAX_IMG_W     1080
#define MAX_IMG_SIZE  (MAX_IMG_H * MAX_IMG_W)

__global__ static void check_visibility_kernel(const VoxelHashTable hash_table, 
                                               const float voxel_size,
                                               const float max_depth,
                                               const CameraParams cam_params,
                                               const SE3<float> cam_P_world,
                                               int *visible_mask) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const VoxelBlock &block = hash_table.GetBlock(idx);
  if (!block.voxels) {
    visible_mask[idx] = 0;
    return;
  }
  const Vector3<float> pos_world = block.block_pos.cast<float>() * (voxel_size * BLOCK_LEN);
  const Vector3<float> pos_cam = cam_P_world.Apply(pos_world);
  const Vector3<float> pos_img_h = cam_params.intrinsics * pos_cam;
  const Vector3<float> pos_img = pos_img_h / pos_img_h.z;
  visible_mask[idx] = (pos_img.x >= 0 && pos_img.x < cam_params.img_w &&
                       pos_img.y >= 0 && pos_img.y < cam_params.img_h &&
                       pos_img_h.z >= 0 && pos_img_h.z < max_depth);
}

__global__ static void gather_visible_blocks_kernel(const VoxelHashTable hash_table,
                                                    const int *visible_mask,
                                                    const int *visible_indics,
                                                    VoxelBlock *output) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (visible_mask[idx]) {
    output[visible_indics[idx] - 1] = hash_table.GetBlock(idx);
  }
}

__global__ static void tsdf_integrate_kernel(VoxelBlock *blocks, 
                                             const SE3<float> cam_P_world,
                                             const CameraParams cam_params,
                                             const int num_visible_blocks,
                                             const float truncation,
                                             const float voxel_size,
                                             const uchar3 *img_rgb,
                                             const float *img_depth,
                                             const float *img_depth_to_range) {
  if (blockIdx.x >= num_visible_blocks) {
    return;
  }
  const Vector3<short> pos_grid_rel(threadIdx.x, threadIdx.y, threadIdx.z);
  // transform to camera / image coordinates
  const Vector3<short> pos_grid_abs = (blocks[blockIdx.x].block_pos << BLOCK_LEN_BITS) 
                                      + pos_grid_rel;
  const Vector3<float> pos_world = pos_grid_abs.cast<float>() * voxel_size;
  const Vector3<float> pos_cam = cam_P_world.Apply(pos_world);
  const Vector3<float> pos_img_h = cam_params.intrinsics * pos_cam;
  const Vector3<float> pos_img = pos_img_h / pos_img_h.z;
  const int u = roundf(pos_img.x);
  const int v = roundf(pos_img.y);
  // update if visible
  if (u >= 0 && u < cam_params.img_w && v >= 0 && v < cam_params.img_h) {
    const int img_idx = v * cam_params.img_w + u;
    const float sdf = img_depth_to_range[img_idx] * (img_depth[img_idx] - pos_img_h.z);
    if (sdf > -truncation) {
      const float tsdf = fminf(1, sdf / truncation);
      const unsigned int idx = offset2index(pos_grid_rel);
      Voxel &voxel = blocks[blockIdx.x].GetVoxelMutable(idx);
      // weight running average
      const float weight_new = 1; // TODO(alvin): add better weighting here
      const float weight_old = voxel.weight;
      const float weight_combined = weight_old + weight_new;
      // rgb running average
      const uchar3 rgb = img_rgb[img_idx];
      const Vector3<float> rgb_old = voxel.rgb.cast<float>();
      const Vector3<float> rgb_new(rgb.x, rgb.y, rgb.z);
      const Vector3<float> rgb_combined = 
        (rgb_old * weight_old + rgb_new * weight_new) / weight_combined;
      voxel.tsdf = (voxel.tsdf * weight_old + tsdf * weight_new) / weight_combined;
      voxel.weight = fminf(roundf(weight_combined), 200); // TODO(alvin): don't hardcode
      voxel.rgb = (rgb_combined + .5).cast<unsigned char>(); // rounding
    }
  }
}

__global__ static void block_allocate_kernel(VoxelHashTable hash_table,
                                             const float *img_depth, 
                                             const CameraParams cam_params,
                                             const SE3<float> world_P_cam,
                                             const float max_depth,
                                             const float truncation,
                                             float *img_depth_to_range) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= cam_params.img_w || y >= cam_params.img_h) {
    return;
  }
  const int idx = y * cam_params.img_w + x;
  const float depth = img_depth[idx];
  if (depth > max_depth) {
    return;
  }
  // transform coordinate from image to world
  const Vector3<float> pos_img_h(x * depth, y * depth, depth);
  const Vector3<float> pos_cam = cam_params.intrinsics_inv * pos_img_h;
  const Vector3<float> pos_world = world_P_cam.Apply(pos_cam);
  // cache depth to range buffer
  const float range = sqrtf(pos_cam.dot(pos_cam));
  img_depth_to_range[idx] = range / depth;
  // calculate end coordinates of sample ray
  const Vector3<float> ray_dir_cam = pos_cam / range;
  const SO3<float> world_R_cam = world_P_cam.GetR();
  const Vector3<float> ray_dir_world = world_R_cam * ray_dir_cam;
  const Vector3<float> ray_start_world = pos_world - ray_dir_world * truncation;
  // put ray into voxel block coordinate
  const Vector3<float> ray_dir_block = ray_dir_world / BLOCK_LEN;
  const Vector3<float> ray_block = 2.f * truncation * ray_dir_world; // start -> end vector
  // DDA for finding ray / block intersection
  const int step = fmaxf(fmaxf(ray_block.x, ray_block.y), ray_block.z);
  const Vector3<float> ray_step_block = ray_block / (float)step;
  Vector3<float> pos_block = ray_start_world / BLOCK_LEN;
  // allocate blocks along the ray
  for (int i = 0; i <= step; ++i, pos_block += ray_step_block) {
    hash_table.Allocate((pos_block + .5).cast<short>());
  }
}

__global__ static void space_carving_kernel(VoxelHashTable hash_table,
                                            const VoxelBlock *blocks, 
                                            const int num_visible_blocks,
                                            const float min_tsdf_threshold) {
  if (blockIdx.x >= num_visible_blocks) {
    return;
  }

  __shared__ float tsdf_abs[BLOCK_VOLUME];
  // load shared buffer
  const int tx = threadIdx.x;
  tsdf_abs[tx] = fabs(blocks[blockIdx.x].GetVoxel(tx).tsdf);
  tsdf_abs[tx + BLOCK_VOLUME/2] = fabs(blocks[blockIdx.x].GetVoxel(tx + BLOCK_VOLUME/2).tsdf);
  // reduce min
  for (int stride = BLOCK_VOLUME/2; stride > 0; stride >>= 1) {
    __syncthreads();
    if (tx < stride) 
      tsdf_abs[tx] = fminf(tsdf_abs[tx], tsdf_abs[tx + stride]);
  }
  // de-allocate block
  if (tx == 0 && tsdf_abs[0] >= min_tsdf_threshold) {
    hash_table.Delete(blocks[blockIdx.x].block_pos);
  }
}

__global__ static void ray_cast_kernel(const VoxelHashTable hash_table,
                                       const CameraParams cam_params,
                                       const SE3<float> world_P_cam,
                                       const float truncation, 
                                       const float max_depth,
                                       const float voxel_size,
                                       float *img_gray) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= cam_params.img_w || y >= cam_params.img_h) {
    return;
  }
  const int idx = y * cam_params.img_w + x;
  const Vector3<float> pos_img_h(x, y, 1);
  const Vector3<float> pos_cam = cam_params.intrinsics_inv * pos_img_h;
  const Vector3<float> ray_dir_cam = pos_cam / sqrtf(pos_cam.dot(pos_cam));
  const SO3<float> world_R_cam = world_P_cam.GetR();
  const Vector3<float> ray_dir_world = world_R_cam * ray_dir_cam;
  const Vector3<float> ray_step_world = ray_dir_world * truncation / 2;
  const int max_step = ceil(max_depth / (truncation / 2));
  Vector3<float> pos_world = world_P_cam.GetT();
  VoxelBlock cache;
  Voxel voxel_prev = hash_table.Retrieve((pos_world / voxel_size + .5).cast<short>(), cache);
  pos_world += ray_step_world;
  for (int i = 1; i < max_step; ++i, pos_world += ray_step_world) {
    const Vector3<short> pos_grid = (pos_world / voxel_size + .5).cast<short>();
    const Voxel voxel_curr = hash_table.Retrieve(pos_grid, cache);
    // ray hit surface
    if (voxel_prev.tsdf > 0 && voxel_curr.tsdf <= 0) {
      Vector3<short> pos1_grid = ((pos_world - ray_step_world) / voxel_size + .5).cast<short>();
      Vector3<short> pos2_grid = pos_grid;
      Vector3<short> pos_mid_grid = (pos1_grid + pos2_grid) / 2;
      // binary search refinement
      while(pos1_grid != pos_mid_grid && pos2_grid != pos_mid_grid) {
        const Voxel voxel = hash_table.Retrieve(pos_mid_grid, cache);
        if (voxel.tsdf < 0) {
          pos2_grid = pos_mid_grid;
        }
        else {
          pos1_grid = pos_mid_grid;
        }
        pos_mid_grid = (pos1_grid + pos2_grid) / 2;
      }
      const Voxel voxel = hash_table.Retrieve(pos_mid_grid, cache);
      // calculate gradient
      const Vector3<float> norm_raw(
        hash_table.Retrieve({ pos2_grid.x + 1, pos2_grid.y, pos2_grid.z }, cache).tsdf - 
        hash_table.Retrieve({ pos2_grid.x - 1, pos2_grid.y, pos2_grid.z }, cache).tsdf,
        hash_table.Retrieve({ pos2_grid.x, pos2_grid.y + 1, pos2_grid.z }, cache).tsdf - 
        hash_table.Retrieve({ pos2_grid.x, pos2_grid.y - 1, pos2_grid.z }, cache).tsdf,
        hash_table.Retrieve({ pos2_grid.x, pos2_grid.y, pos2_grid.z + 1 }, cache).tsdf - 
        hash_table.Retrieve({ pos2_grid.x, pos2_grid.y, pos2_grid.z - 1 }, cache).tsdf
      );
      img_gray[idx] = -fminf(norm_raw.dot(ray_dir_world) / sqrtf(norm_raw.dot(norm_raw)), 0);
      return;
    }
    voxel_prev = voxel_curr;
  }
  img_gray[idx] = 0; // no surface intersection found
}

TSDFGrid::TSDFGrid(float voxel_size, float truncation, float max_depth) 
  : voxel_size_(voxel_size), truncation_(truncation), max_depth_(max_depth) {
  // memory allocation
  CUDA_SAFE_CALL(hipMalloc(&visible_mask_, sizeof(int) * NUM_ENTRY));
  CUDA_SAFE_CALL(hipMalloc(&visible_indics_, sizeof(int) * NUM_ENTRY));
  CUDA_SAFE_CALL(hipMalloc(&visible_indics_aux_, sizeof(int) * SCAN_BLOCK_SIZE));
  CUDA_SAFE_CALL(hipMalloc(&visible_blocks_, sizeof(VoxelBlock) * NUM_ENTRY));
  CUDA_SAFE_CALL(hipMalloc(&img_rgb_, sizeof(uint3) * MAX_IMG_SIZE));
  CUDA_SAFE_CALL(hipMalloc(&img_depth_, sizeof(float) * MAX_IMG_SIZE));
  CUDA_SAFE_CALL(hipMalloc(&img_depth_to_range_, sizeof(float) * MAX_IMG_SIZE));
  CUDA_SAFE_CALL(hipMalloc(&img_normal_, sizeof(float) * MAX_IMG_SIZE));
  // stream init
  CUDA_SAFE_CALL(hipStreamCreate(&stream_));
}

TSDFGrid::~TSDFGrid() {
  // release memory
  hash_table_.ReleaseMemory();
  CUDA_SAFE_CALL(hipFree(visible_mask_));
  CUDA_SAFE_CALL(hipFree(visible_indics_));
  CUDA_SAFE_CALL(hipFree(visible_indics_aux_));
  CUDA_SAFE_CALL(hipFree(visible_blocks_));
  CUDA_SAFE_CALL(hipFree(img_rgb_));
  CUDA_SAFE_CALL(hipFree(img_depth_));
  CUDA_SAFE_CALL(hipFree(img_depth_to_range_));
  CUDA_SAFE_CALL(hipFree(img_normal_));
  // release cuda stream
  CUDA_SAFE_CALL(hipStreamDestroy(stream_));
}

void TSDFGrid::Integrate(const cv::Mat &img_rgb, const cv::Mat &img_depth, 
                         const CameraIntrinsics<float> &intrinsics, 
                         const SE3<float> &cam_P_world) {
  assert(img_rgb.type() == CV_8UC3);
  assert(img_depth.type() == CV_32FC1);
  assert(img_rgb.cols == img_depth.cols);
  assert(img_rgb.rows == img_depth.rows);

  const CameraParams cam_params(intrinsics, img_rgb.rows, img_rgb.cols);

  Allocate(img_rgb, img_depth, cam_params, cam_P_world);
  const int num_visible_blocks = GatherVisible(cam_params, cam_P_world);
  UpdateTSDF(num_visible_blocks, cam_params, cam_P_world);
  SpaceCarving(num_visible_blocks);
}

void TSDFGrid::Allocate(const cv::Mat &img_rgb, const cv::Mat &img_depth,
                        const CameraParams &cam_params, const SE3<float> &cam_P_world) {
  CUDA_SAFE_CALL(hipMemcpyAsync(img_rgb_, img_rgb.data, 
    sizeof(char)*img_rgb.total(), hipMemcpyHostToDevice, stream_));
  CUDA_SAFE_CALL(hipMemcpyAsync(img_depth_, img_depth.data, 
    sizeof(float)*img_depth.total(), hipMemcpyHostToDevice, stream_));
  const dim3 IMG_BLOCK_DIM(ceil((float)cam_params.img_w/32), ceil((float)cam_params.img_h/16));
  const dim3 IMG_THREAD_DIM(32, 16);
  block_allocate_kernel<<<IMG_BLOCK_DIM, IMG_THREAD_DIM, 0, stream_>>>(
    hash_table_, img_depth_, cam_params, cam_P_world.Inverse(), 
    max_depth_, truncation_, img_depth_to_range_);
  hash_table_.ResetLocks(stream_);
}

int TSDFGrid::GatherVisible(const CameraParams &cam_params, const SE3<float> &cam_P_world) {
  constexpr int GATHER_THREAD_DIM = 512;
  const int GATHER_BLOCK_DIM = ceil((float)NUM_ENTRY / GATHER_THREAD_DIM);
  // generate binary array of visibility
  check_visibility_kernel<<<GATHER_BLOCK_DIM, GATHER_THREAD_DIM, 0, stream_>>>(
    hash_table_, voxel_size_, max_depth_, cam_params,
    cam_P_world, visible_mask_);
  // parallel prefix sum scan
  prefix_sum<int>(visible_mask_, visible_indics_, visible_indics_aux_, NUM_ENTRY, stream_);
  // gather visible blocks into contiguous array
  gather_visible_blocks_kernel<<<GATHER_BLOCK_DIM, GATHER_THREAD_DIM, 0, stream_>>>(
    hash_table_, visible_mask_, visible_indics_, visible_blocks_);
  int num_visible_blocks;
  CUDA_SAFE_CALL(hipMemcpyAsync(&num_visible_blocks, visible_indics_ + NUM_ENTRY - 1,
    sizeof(int), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipStreamSynchronize(stream_));
  return num_visible_blocks;
}

void TSDFGrid::UpdateTSDF(int num_visible_blocks, 
                          const CameraParams &cam_params, const SE3<float> &cam_P_world) {
  const dim3 VOXEL_BLOCK_DIM(BLOCK_LEN, BLOCK_LEN, BLOCK_LEN);
  tsdf_integrate_kernel<<<num_visible_blocks, VOXEL_BLOCK_DIM, 0, stream_>>>(
    visible_blocks_, cam_P_world, cam_params, num_visible_blocks, truncation_, voxel_size_,
    img_rgb_, img_depth_, img_depth_to_range_);
}

void TSDFGrid::SpaceCarving(int num_visible_blocks) {
  space_carving_kernel<<<num_visible_blocks, BLOCK_VOLUME/2, 0, stream_>>>(
    hash_table_, visible_blocks_, num_visible_blocks, .8); 
  hash_table_.ResetLocks(stream_);
}

void TSDFGrid::RayCast(cv::Mat *img, 
                       const CameraIntrinsics<float> &virtual_intrinsics,
                       const SE3<float> &cam_P_world) {
  assert(img->type() == CV_32FC1);

  const CameraParams cam_params(virtual_intrinsics, img->rows, img->cols);
  const dim3 IMG_BLOCK_DIM(ceil((float)cam_params.img_w/32), ceil((float)cam_params.img_h/16));
  const dim3 IMG_THREAD_DIM(32, 16);
  ray_cast_kernel<<<IMG_BLOCK_DIM, IMG_THREAD_DIM, 0, stream_>>>(
    hash_table_, cam_params, cam_P_world.Inverse(),
    truncation_, max_depth_, voxel_size_, img_normal_);
  CUDA_SAFE_CALL(hipMemcpyAsync(img->data, img_normal_, 
    sizeof(float) * img->total(), hipMemcpyDeviceToHost, stream_));
  CUDA_SAFE_CALL(hipStreamSynchronize(stream_));
}
