#include "hip/hip_runtime.h"
#include <cassert>

#include "utils/cuda/arithmetic.cuh"
#include "utils/cuda/errors.cuh"
#include "utils/tsdf/voxel_hash.cuh"

__global__ static void reset_locks_kernel(int *locks, int num_locks) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_locks) {
    locks[idx] = FREE;
  }
}

__device__ __host__ uint hash(const Vector3<short> &block_pos) {
  return (((uint)block_pos.x * 73856093u) ^
          ((uint)block_pos.y * 19349669u) ^
          ((uint)block_pos.z * 83492791u)) & BUCKET_MASK; 
}

__device__ __host__ VoxelBlock::VoxelBlock() {
  this->offset = 0;
  this->voxels = NULL;
}

__device__ __host__ Voxel VoxelBlock::GetVoxel(const Vector3<short> &point) const {
  return GetVoxelMutable(point);
}

__device__ __host__ Voxel VoxelBlock::GetVoxel(const short idx) const {
  return GetVoxelMutable(idx);
}

__device__ __host__ Voxel& VoxelBlock::GetVoxelMutable(const Vector3<short> &point) const {
  assert(voxels != NULL);
  assert(point2block(point) == this->block_pos);
  const Vector3<short> offset = point2offset(point);
  const unsigned short idx = offset2index(offset); 
  return voxels[idx];
}

__device__ __host__ Voxel& VoxelBlock::GetVoxelMutable(const short idx) const {
  assert(voxels != NULL);
  assert(idx < BLOCK_VOLUME);
  return voxels[idx];
}

VoxelHashTable::VoxelHashTable() : VoxelMemPool() {
  // initialize hash table
  CUDA_SAFE_CALL(hipMalloc(&hash_table_, sizeof(VoxelBlock) * NUM_ENTRY));
  CUDA_SAFE_CALL(hipMemset(hash_table_, 0, sizeof(VoxelBlock) * NUM_ENTRY));
  // initialize bucket locks
  CUDA_SAFE_CALL(hipMalloc(&bucket_locks_, sizeof(BucketLock) * NUM_BUCKET));
  CUDA_SAFE_CALL(hipMemset(bucket_locks_, FREE, sizeof(BucketLock) * NUM_BUCKET));
}

void VoxelHashTable::ResetLocks(hipStream_t stream) {
  reset_locks_kernel<<<NUM_BUCKET / 1024, 1024, 0, stream>>>(bucket_locks_, NUM_BUCKET);
  CUDA_STREAM_CHECK_ERROR(stream);
}

void VoxelHashTable::ReleaseMemory() {
  CUDA_SAFE_CALL(hipFree(hash_table_));
  CUDA_SAFE_CALL(hipFree(bucket_locks_));
  VoxelMemPool::ReleaseMemory();
}

__device__ void VoxelHashTable::Allocate(const Vector3<short> &block_pos) {
  const unsigned int bucket_idx = hash(block_pos);
  const unsigned int entry_idx = (bucket_idx << NUM_ENTRY_PER_BUCKET_BITS);
  // check for current bucket
  #pragma unroll
  for(int i = 0; i < NUM_ENTRY_PER_BUCKET; ++i) {
    VoxelBlock &block = hash_table_[entry_idx + i];
    if (block.block_pos == block_pos && block.voxels) { return; }
    if (block.voxels == NULL) {
      if (atomicExch(&bucket_locks_[bucket_idx], LOCKED) == FREE) { // lock bucket
        block.block_pos = block_pos;
        block.offset = 0;
        block.voxels = this->AquireBlock();
      }
      return;
    }
  }
  // traverse list
  unsigned int entry_idx_last = entry_idx + NUM_ENTRY_PER_BUCKET - 1;
  while (hash_table_[entry_idx_last].offset) {
    entry_idx_last = (entry_idx_last + hash_table_[entry_idx_last].offset) & ENTRY_MASK;
    const VoxelBlock &block = hash_table_[entry_idx_last];
    if (block.block_pos == block_pos && block.voxels) { return; }
  }
  const unsigned int bucket_idx_last = entry_idx_last >> NUM_ENTRY_PER_BUCKET_BITS;
  // append to list
  unsigned int entry_idx_next = entry_idx_last;
  while (true) {
    entry_idx_next = (entry_idx_next + 1) & ENTRY_MASK;
    // not last position of the bucket && hash entry empty
    if ((entry_idx_next & ENTRY_PER_BUCKET_MASK) != ENTRY_PER_BUCKET_MASK &&
        hash_table_[entry_idx_next].voxels == NULL) { 
      if (atomicExch(&bucket_locks_[bucket_idx_last], LOCKED) == FREE) { // lock last bucket
        const unsigned int bucket_idx_next = entry_idx_next >> NUM_ENTRY_PER_BUCKET_BITS;
        if (atomicExch(&bucket_locks_[bucket_idx_next], LOCKED) == FREE) { // lock next bucket
          VoxelBlock &block_last = hash_table_[entry_idx_last];
          VoxelBlock &block_next = hash_table_[entry_idx_next];
          // link new node to previous list tail
          const unsigned int wrap = entry_idx_next > entry_idx_last ? 0 : NUM_ENTRY;
          block_last.offset = entry_idx_next + wrap - entry_idx_last;
          // allocate new hash entry
          block_next.block_pos = block_pos;
          block_next.offset = 0;
          block_next.voxels = this->AquireBlock();
        }
      }
      return;
    }
  }
}

__device__ void VoxelHashTable::Delete(const Vector3<short> &block_pos) {
  const unsigned int bucket_idx = hash(block_pos);
  const unsigned int entry_idx = (bucket_idx << NUM_ENTRY_PER_BUCKET_BITS);
  // check for current bucket
  #pragma unroll
  for(int i = 0; i < NUM_ENTRY_PER_BUCKET - 1; ++i) {
    VoxelBlock &block = hash_table_[entry_idx + i];
    if (block.block_pos == block_pos && block.voxels) { 
      this->ReleaseBlock(block.voxels);
      block.offset = 0;
      block.voxels = NULL;
      return; 
    }
  }
  // special handling for list head
  unsigned int entry_idx_last = entry_idx + NUM_ENTRY_PER_BUCKET - 1;
  VoxelBlock &block_head = hash_table_[entry_idx_last];
  if (block_head.block_pos == block_pos && block_head.voxels) {
    if (atomicExch(&bucket_locks_[bucket_idx], LOCKED) == FREE) {
      const unsigned int entry_idx_next = (entry_idx_last + block_head.offset) & ENTRY_MASK;
      VoxelBlock &block_next = hash_table_[entry_idx_next];
      this->ReleaseBlock(block_head.voxels);
      block_head.block_pos = block_next.block_pos;
      // check if reaches tail
      block_head.offset = block_next.offset ? block_head.offset + block_next.offset : 0;
      block_head.voxels = block_next.voxels;
      block_next.offset = 0;
      block_next.voxels = NULL;
    }
    return;
  }
  // generic list handling
  while (hash_table_[entry_idx_last].offset) {
    VoxelBlock &block_last = hash_table_[entry_idx_last];
    const unsigned int entry_idx_curr = (entry_idx_last + block_last.offset) & ENTRY_MASK;
    VoxelBlock &block_curr = hash_table_[entry_idx_curr];
    if (block_curr.block_pos == block_pos && block_curr.voxels) {
      if (atomicExch(&bucket_locks_[bucket_idx], LOCKED) == FREE) { // lock original bucket
        // check if reaches tail
        block_last.offset = block_curr.offset ? block_last.offset + block_curr.offset : 0;
        // free current entry
        this->ReleaseBlock(block_curr.voxels);
        block_curr.offset = 0;
        block_curr.voxels = NULL;
      }
      return;
    }
    entry_idx_last = entry_idx_curr;
  }
}

__device__ float VoxelHashTable::RetrieveTSDF(const Vector3<float> &point, 
                                              VoxelBlock &cache) const {
  const Vector3<float> point_l = point.cast<short>().cast<float>();
  const Vector3<float> point_h = point_l + 1;
  const Vector3<float> alpha = point_h - point;
  const float tsdf_000 = Retrieve(Vector3<short>(point_l.x, point_l.y, point_l.z), cache).tsdf;
  const float tsdf_001 = Retrieve(Vector3<short>(point_h.x, point_l.y, point_h.z), cache).tsdf;
  const float tsdf_010 = Retrieve(Vector3<short>(point_l.x, point_h.y, point_l.z), cache).tsdf;
  const float tsdf_011 = Retrieve(Vector3<short>(point_l.x, point_h.y, point_h.z), cache).tsdf;
  const float tsdf_100 = Retrieve(Vector3<short>(point_h.x, point_l.y, point_l.z), cache).tsdf;
  const float tsdf_101 = Retrieve(Vector3<short>(point_h.x, point_l.y, point_h.z), cache).tsdf;
  const float tsdf_110 = Retrieve(Vector3<short>(point_h.x, point_h.y, point_l.z), cache).tsdf;
  const float tsdf_111 = Retrieve(Vector3<short>(point_h.x, point_h.y, point_h.z), cache).tsdf;
  // interpolate across z
  const float tsdf_00 = tsdf_000 * alpha.z + tsdf_001 * (1 - alpha.z);
  const float tsdf_01 = tsdf_010 * alpha.z + tsdf_011 * (1 - alpha.z);
  const float tsdf_10 = tsdf_100 * alpha.z + tsdf_101 * (1 - alpha.z);
  const float tsdf_11 = tsdf_110 * alpha.z + tsdf_111 * (1 - alpha.z);
  // interpolate across y
  const float tsdf_0 = tsdf_00 * alpha.y + tsdf_01 * (1 - alpha.y);
  const float tsdf_1 = tsdf_10 * alpha.y + tsdf_11 * (1 - alpha.y);
  // interpolate across x
  return tsdf_0 * alpha.x + tsdf_1 * (1 - alpha.x);
}

__device__ Voxel VoxelHashTable::Retrieve(const Vector3<short> &point, VoxelBlock &cache) const {
  Voxel *voxel = RetrieveMutable(point, cache);
  if (voxel)
    return *voxel;
  // not found -> empty space
  return { 1., { 0, 0, 0 }, 0 };
}

__device__ Voxel* VoxelHashTable::RetrieveMutable(const Vector3<short> &point, 
                                                  VoxelBlock &cache) const {
  const Vector3<short> block_pos = point2block(point);
  if (cache.block_pos == block_pos) {
    if (cache.voxels) {
      return &(cache.GetVoxelMutable(point));
    }
    else if (cache.offset < 0) {
      return NULL;
    }
  }
  const unsigned int bucket_idx = hash(block_pos);
  const unsigned int entry_idx = (bucket_idx << NUM_ENTRY_PER_BUCKET_BITS);
  // check for current bucket
  #pragma unroll
  for(int i = 0; i < NUM_ENTRY_PER_BUCKET; ++i) {
    VoxelBlock &block = hash_table_[entry_idx + i];
    if (block.block_pos == block_pos && block.voxels) { 
      cache = block;
      return &(cache.GetVoxelMutable(point));
    }
  }
  // traverse list
  unsigned int entry_idx_last = entry_idx + NUM_ENTRY_PER_BUCKET - 1;
  while (hash_table_[entry_idx_last].offset) {
    entry_idx_last = (entry_idx_last + hash_table_[entry_idx_last].offset) & ENTRY_MASK;
    const VoxelBlock &block = hash_table_[entry_idx_last];
    if (block.block_pos == block_pos && block.voxels) { 
      cache = block;
      return &(cache.GetVoxelMutable(point));
    }
  }
  // not found
  cache.block_pos = block_pos;
  cache.offset = -1;
  cache.voxels = NULL;
  return NULL;
}

__device__ const VoxelBlock& VoxelHashTable::GetBlock(const int idx) const {
  assert(idx < NUM_ENTRY);
  return hash_table_[idx];
}

__device__ __host__ int VoxelHashTable::NumActiveBlock() const {
  return NUM_BLOCK - *num_free_blocks_;
}
