#include "hip/hip_runtime.h"
#include <cassert>
#include <gtest/gtest.h>

#include "utils/cuda/errors.cuh"
#include "utils/tsdf/voxel_hash.cuh"

#define MAX_BLOCKS 128

class VoxelHashTest : public ::testing::Test {
 protected:
  VoxelHashTest() {
    CUDA_SAFE_CALL(hipMallocManaged(&voxel, sizeof(Voxel) * MAX_BLOCKS * BLOCK_VOLUME));
    CUDA_SAFE_CALL(hipMallocManaged(&voxel_block, sizeof(VoxelBlock) * MAX_BLOCKS));
    CUDA_SAFE_CALL(hipMallocManaged(&point, sizeof(Vector3<short>) * MAX_BLOCKS * BLOCK_VOLUME));
    CUDA_SAFE_CALL(hipMallocManaged(&block_pos, sizeof(Vector3<short>) * MAX_BLOCKS));
  }

  ~VoxelHashTest() {
    voxel_hash_table.ReleaseMemory();
    CUDA_SAFE_CALL(hipFree(voxel));
    CUDA_SAFE_CALL(hipFree(voxel_block));
    CUDA_SAFE_CALL(hipFree(point));
    CUDA_SAFE_CALL(hipFree(block_pos));
  }

  VoxelHashTable voxel_hash_table;
  Voxel *voxel;
  VoxelBlock *voxel_block;
  Vector3<short> *point;
  Vector3<short> *block_pos;
};

__global__ void Allocate(VoxelHashTable hash_table, Vector3<short> *block_pos) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hash_table.Allocate(block_pos[idx]);
}

__global__ void Retrieve(VoxelHashTable hash_table, 
                         const Vector3<short> *point, Voxel *voxel, VoxelBlock *voxel_block) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  voxel[idx] = hash_table.Retrieve(point[idx], voxel_block[idx]);
}

__global__ void Assignment(VoxelHashTable hash_table, const Vector3<short> *point, Voxel *voxel) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  VoxelBlock block;
  Voxel *voxel_old = hash_table.RetrieveMutable(point[idx], block);
  assert(voxel_old != NULL);
  *voxel_old = voxel[idx];
}


TEST_F(VoxelHashTest, Single) {
  // allocate block (1, 1, 1)
  *block_pos = Vector3<short>(1);
  *point = Vector3<short>(8);
  Allocate<<<1, 1>>>(voxel_hash_table, block_pos);
  CUDA_CHECK_ERROR;
  Retrieve<<<1, 1>>>(voxel_hash_table, point, voxel, voxel_block);
  CUDA_SAFE_DEVICE_SYNC;
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 1);
  EXPECT_EQ(voxel_block->block_pos, *block_pos);
  // retrieve empty block
  *point = Vector3<short>(0);
  Retrieve<<<1, 1>>>(voxel_hash_table, point, voxel, voxel_block);
  CUDA_SAFE_DEVICE_SYNC;
  EXPECT_EQ(voxel->weight, 0);
  // assignment
  *block_pos = Vector3<short>(0);
  Allocate<<<1, 1>>>(voxel_hash_table, block_pos);
  CUDA_CHECK_ERROR;
  voxel_block->offset = 0; // reset cache after re allocation
  for (unsigned char i = 0; i < BLOCK_LEN; ++i) {
    *point = { 0, 0, i };
    *voxel = { 1, { i, i, i }, i };
    Assignment<<<1, 1>>>(voxel_hash_table, point, voxel);
    CUDA_SAFE_DEVICE_SYNC;
  }
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 2);
  for (unsigned char i = 0; i < BLOCK_LEN; ++i) {
    *point = { 0, 0, i };
    Retrieve<<<1, 1>>>(voxel_hash_table, point, voxel, voxel_block);
    CUDA_SAFE_DEVICE_SYNC;
    EXPECT_EQ(voxel->sdf, 1);
    EXPECT_EQ(voxel->rgb[0], i);
    EXPECT_EQ(voxel->rgb[1], i);
    EXPECT_EQ(voxel->rgb[2], i);
    EXPECT_EQ(voxel->weight, i);
  }
}

TEST_F(VoxelHashTest, Multiple) {
  for (unsigned char i = 0; i < MAX_BLOCKS; ++i) {
    block_pos[i] = { i, i, i };
  }
  Allocate<<<1, MAX_BLOCKS>>>(voxel_hash_table, block_pos);
  voxel_hash_table.ResetLocks();
  CUDA_SAFE_DEVICE_SYNC;
  // check received (assume no collision)
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), MAX_BLOCKS);
  // assign some voxels
  for (unsigned char i = 0; i < MAX_BLOCKS; ++i) {
    point[i] = Vector3<short>(i * BLOCK_LEN);
    voxel[i] = { 1, { i, i, i }, i };
  }
  Assignment<<<1, MAX_BLOCKS>>>(voxel_hash_table, point, voxel);
  CUDA_SAFE_DEVICE_SYNC;
  // reset buffer
  for (unsigned char i = 0; i < MAX_BLOCKS; ++i) {
    voxel[i] = { 0, { 0, 0, 0 }, 0 };
    block_pos[i] = { 0, 0, 0 };
  }
  // retrieve and verify
  Retrieve<<<1, MAX_BLOCKS>>>(voxel_hash_table, point, voxel, voxel_block);
  CUDA_SAFE_DEVICE_SYNC;
  for (unsigned char i = 0; i < MAX_BLOCKS; ++i) {
    EXPECT_EQ(voxel[i].sdf, 1);
    EXPECT_EQ(voxel[i].rgb[0], i);
    EXPECT_EQ(voxel[i].rgb[1], i);
    EXPECT_EQ(voxel[i].rgb[2], i);
    EXPECT_EQ(voxel[i].weight, i);
    EXPECT_EQ(voxel_block[i].block_pos, Vector3<short>(i));
  }
} 

TEST_F(VoxelHashTest, Collision) {
  // all hash to the last index NUM_BUCKET - 1
  block_pos[0] = { 33, 180, 42 };
  block_pos[1] = { 61, 16, 170 };
  block_pos[2] = { 63, 171, 45 };
  ASSERT_EQ(hash(block_pos[0]), NUM_BUCKET - 1);
  ASSERT_EQ(hash(block_pos[0]), hash(block_pos[1]));
  ASSERT_EQ(hash(block_pos[1]), hash(block_pos[2]));
  // hash to another idx
  block_pos[3] = { 0, 0, 0 };
  // allocate with conflict
  Allocate<<<1, 4>>>(voxel_hash_table, block_pos);
  voxel_hash_table.ResetLocks();
  CUDA_SAFE_DEVICE_SYNC;
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 2);
  // allocate again
  Allocate<<<1, 4>>>(voxel_hash_table, block_pos);
  voxel_hash_table.ResetLocks();
  CUDA_SAFE_DEVICE_SYNC;
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 3);
  // allocate yet again
  Allocate<<<1, 4>>>(voxel_hash_table, block_pos);
  voxel_hash_table.ResetLocks();
  CUDA_SAFE_DEVICE_SYNC;
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 4);
  // do some assignment
  for (unsigned char i = 0; i < 4; ++i) {
    point[i] = block_pos[i] * BLOCK_LEN; // use the first point of a block
    voxel[i] = { 1, { i, i, i }, i };
  }
  Assignment<<<1, 4>>>(voxel_hash_table, point, voxel);
  CUDA_SAFE_DEVICE_SYNC;
  // reset buffer
  for (unsigned char i = 0; i < 4; ++i) {
    voxel[i] = { 0, { 0, 0, 0 }, 0 };
    block_pos[i] = { 0, 0, 0 };
  }
  // retrieve and verify
  Retrieve<<<1, 4>>>(voxel_hash_table, point, voxel, voxel_block);
  CUDA_SAFE_DEVICE_SYNC;
  for (unsigned char i = 0; i < 4; ++i) {
    EXPECT_EQ(voxel[i].sdf, 1);
    EXPECT_EQ(voxel[i].rgb[0], i);
    EXPECT_EQ(voxel[i].rgb[1], i);
    EXPECT_EQ(voxel[i].rgb[2], i);
    EXPECT_EQ(voxel[i].weight, i);
  }
}
