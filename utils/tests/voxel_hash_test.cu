#include "hip/hip_runtime.h"
#include <cassert>
#include <gtest/gtest.h>

#include "utils/tsdf/voxel_hash.cuh"

#define MAX_BLOCKS 128

class VoxelHashTest : public ::testing::Test {
 protected:
  VoxelHashTest() {
    hipMallocManaged(&voxel, sizeof(Voxel) * MAX_BLOCKS * BLOCK_VOLUME);
    hipMallocManaged(&voxel_block, sizeof(VoxelBlock) * MAX_BLOCKS);
    hipMallocManaged(&point, sizeof(Vector3<short>) * MAX_BLOCKS * BLOCK_VOLUME);
    hipMallocManaged(&block_pos, sizeof(Vector3<short>) * MAX_BLOCKS);
  }

  ~VoxelHashTest() {
    voxel_hash_table.ReleaseMemory();
    hipFree(voxel);
    hipFree(voxel_block);
    hipFree(point);
    hipFree(block_pos);
  }

  VoxelHashTable voxel_hash_table;
  Voxel *voxel;
  VoxelBlock *voxel_block;
  Vector3<short> *point;
  Vector3<short> *block_pos;
};

__global__ void Allocate(VoxelHashTable hash_table, Vector3<short> *block_pos) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hash_table.Allocate(block_pos[idx]);
}

__global__ void Retrieve(VoxelHashTable hash_table, 
                         const Vector3<short> *point, Voxel *voxel, VoxelBlock *voxel_block) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  voxel[idx] = hash_table.Retrieve(point[idx], voxel_block[idx]);
}

__global__ void Assignment(VoxelHashTable hash_table, const Vector3<short> *point, Voxel *voxel) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  VoxelBlock block;
  Voxel *voxel_old = hash_table.RetrieveMutable(point[idx], block);
  assert(voxel_old != NULL);
  *voxel_old = voxel[idx];
}


TEST_F(VoxelHashTest, Single) {
  // allocate block (1, 1, 1)
  *block_pos = Vector3<short>(1);
  *point = Vector3<short>(8);
  Allocate<<<1, 1>>>(voxel_hash_table, block_pos);
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  Retrieve<<<1, 1>>>(voxel_hash_table, point, voxel, voxel_block);
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  hipDeviceSynchronize();
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 1);
  EXPECT_EQ(voxel_block->block_pos, *block_pos);
  // retrieve empty block
  *point = Vector3<short>(0);
  Retrieve<<<1, 1>>>(voxel_hash_table, point, voxel, voxel_block);
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  hipDeviceSynchronize();
  EXPECT_EQ(voxel->weight, 0);
  // assignment
  *block_pos = Vector3<short>(0);
  Allocate<<<1, 1>>>(voxel_hash_table, block_pos);
  voxel_block->offset = 0; // reset cache after re allocation
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (unsigned char i = 0; i < BLOCK_LEN; ++i) {
    *point = { 0, 0, i };
    *voxel = { 1, { i, i, i }, i };
    Assignment<<<1, 1>>>(voxel_hash_table, point, voxel);
    hipDeviceSynchronize();
  }
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 2);
  for (unsigned char i = 0; i < BLOCK_LEN; ++i) {
    *point = { 0, 0, i };
    Retrieve<<<1, 1>>>(voxel_hash_table, point, voxel, voxel_block);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
    hipDeviceSynchronize();
    EXPECT_EQ(voxel->sdf, 1);
    EXPECT_EQ(voxel->rgb[0], i);
    EXPECT_EQ(voxel->rgb[1], i);
    EXPECT_EQ(voxel->rgb[2], i);
    EXPECT_EQ(voxel->weight, i);
  }
}

TEST_F(VoxelHashTest, Multiple) {
  for (unsigned char i = 0; i < MAX_BLOCKS; ++i) {
    block_pos[i] = { i, i, i };
  }
  Allocate<<<1, MAX_BLOCKS>>>(voxel_hash_table, block_pos);
  voxel_hash_table.ResetLocks();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  // check received (assume no collision)
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), MAX_BLOCKS);
  // assign some voxels
  for (unsigned char i = 0; i < MAX_BLOCKS; ++i) {
    point[i] = Vector3<short>(i * BLOCK_LEN);
    voxel[i] = { 1, { i, i, i }, i };
  }
  Assignment<<<1, MAX_BLOCKS>>>(voxel_hash_table, point, voxel);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  // reset buffer
  for (unsigned char i = 0; i < MAX_BLOCKS; ++i) {
    voxel[i] = { 0, { 0, 0, 0 }, 0 };
    block_pos[i] = { 0, 0, 0 };
  }
  // retrieve and verify
  Retrieve<<<1, MAX_BLOCKS>>>(voxel_hash_table, point, voxel, voxel_block);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (unsigned char i = 0; i < MAX_BLOCKS; ++i) {
    EXPECT_EQ(voxel[i].sdf, 1);
    EXPECT_EQ(voxel[i].rgb[0], i);
    EXPECT_EQ(voxel[i].rgb[1], i);
    EXPECT_EQ(voxel[i].rgb[2], i);
    EXPECT_EQ(voxel[i].weight, i);
    EXPECT_EQ(voxel_block[i].block_pos, Vector3<short>(i));
  }
} 

TEST_F(VoxelHashTest, Collision) {
  // all hash to the very last index (1 << NUM_BUCKET_BITS) - 1
  block_pos[0] = { 33, 180, 42 };
  block_pos[1] = { 61, 16, 170 };
  block_pos[2] = { 63, 171, 45 };
  ASSERT_EQ(hash(block_pos[0]), hash(block_pos[1]));
  ASSERT_EQ(hash(block_pos[1]), hash(block_pos[2]));
  // hash to another idx
  block_pos[3] = { 0, 0, 0 };
  // allocate with conflict
  Allocate<<<1, 4>>>(voxel_hash_table, block_pos);
  voxel_hash_table.ResetLocks();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 2);
  // allocate again
  Allocate<<<1, 4>>>(voxel_hash_table, block_pos);
  voxel_hash_table.ResetLocks();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 3);
  // allocate yet again
  Allocate<<<1, 4>>>(voxel_hash_table, block_pos);
  voxel_hash_table.ResetLocks();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  EXPECT_EQ(voxel_hash_table.NumActiveBlock(), 4);
  // do some assignment
  for (unsigned char i = 0; i < 4; ++i) {
    point[i] = block_pos[i] * BLOCK_LEN; // use the first point of a block
    voxel[i] = { 1, { i, i, i }, i };
  }
  Assignment<<<1, 4>>>(voxel_hash_table, point, voxel);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  // reset buffer
  for (unsigned char i = 0; i < 4; ++i) {
    voxel[i] = { 0, { 0, 0, 0 }, 0 };
    block_pos[i] = { 0, 0, 0 };
  }
  // retrieve and verify
  Retrieve<<<1, 4>>>(voxel_hash_table, point, voxel, voxel_block);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (unsigned char i = 0; i < 4; ++i) {
    EXPECT_EQ(voxel[i].sdf, 1);
    EXPECT_EQ(voxel[i].rgb[0], i);
    EXPECT_EQ(voxel[i].rgb[1], i);
    EXPECT_EQ(voxel[i].rgb[2], i);
    EXPECT_EQ(voxel[i].weight, i);
  }
}
